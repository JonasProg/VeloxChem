//
//                     V.E.L.O.X. C.H.E.M. MP
//      ---------------------------------------------------
//           An Electronic Structure Code for Nanoscale
//
//  Created by Zilvinas Rinkevicius (rinkevic@kth.se), KTH, Sweden.
//  Copyright © 2018 by Velox Chem MP developers. All rights reserved.

#include "DeviceFunc.hpp"

#include "ErrorHandler.hpp"

namespace gpu {  // gpu namespace

void
set_device(const int32_t iDevice)
{
#ifdef ENABLE_GPU
    auto cerr = hipSetDevice(iDevice);

    errors::assertMsgCritical(cerr == hipSuccess, {"setCudaDevice"});
#endif
}

void
synchronize_device()
{
#ifdef ENABLE_GPU
auto cerr = hipDeviceSynchronize();

errors::assertMsgCritical(cerr == hipSuccess, {"synchronizeCudaDevice"});
#endif
}

void
allocateDeviceMemory(void**  pointer,
                     size_t* dataPitch,
                     size_t  dataWidth,
                     size_t  dataHeight)
{
#ifdef ENABLE_GPU
    auto cerr = hipMallocPitch(pointer, dataPitch, dataWidth, dataHeight);

    errors::assertMsgCritical(cerr == hipSuccess, {"allocateDeviceMemory"});
#endif
}

void
freeDeviceMemory(void* pointerToMemory)
{
#ifdef ENABLE_GPU
    auto cerr = hipFree(pointerToMemory);

    errors::assertMsgCritical(cerr == hipSuccess, {"freeDeviceMemory"});
#endif
}

void
copyToDeviceMemory(      void*  destination,
                         size_t destinationPitch,
                   const void*  source,
                         size_t sourcePitch,
                         size_t dataWidth,
                         size_t dataHeight)
{
#ifdef ENABLE_GPU
    auto cerr = hipMemcpy2D(destination, destinationPitch, source, sourcePitch, dataWidth, dataHeight,
                             hipMemcpyHostToDevice);

    errors::assertMsgCritical(cerr == hipSuccess, {"copyToDeviceMemory"});
#endif
}

void
copyFromDeviceMemory(     void*  destination,
                          size_t destinationPitch,
                    const void*  source,
                          size_t sourcePitch,
                          size_t dataWidth,
                          size_t dataHeight)
{
#ifdef ENABLE_GPU
    auto cerr = hipMemcpy2D(destination, destinationPitch, source, sourcePitch, dataWidth, dataHeight,
                             hipMemcpyDeviceToHost);

    errors::assertMsgCritical(cerr == hipSuccess, {"copyFromDeviceMemory"});
#endif
}

}  // namespace gpu
