//
//                     V.E.L.O.X. C.H.E.M. MP
//      ---------------------------------------------------
//           An Electronic Structure Code for Nanoscale
//
//  Created by Zilvinas Rinkevicius (rinkevic@kth.se), KTH, Sweden.
//  Copyright © 2018 by Velox Chem MP developers. All rights reserved.

#include "DeviceProp.hpp"
 
#include <cmath>
#include <sstream>
#include <string>

#include "StringFormat.hpp"

namespace gpu {  // gpu namespace

void
get_devices_property(std::vector<std::string>& namesOfDevices,
                    std::vector<int32_t>&     globalMemoryOfDevices,
                    std::vector<int32_t>&     computeMajorCapabilityOfDevices,
                    std::vector<int32_t>&     computeMinorCapabilityOfDevices)
{
#ifdef ENABLE_GPU

    int devcnt = 0;

    hipGetDeviceCount(&devcnt);

    for (int i = 0; i < devcnt; i++)
    {
        hipDeviceProp_t prop;

        hipGetDeviceProperties(&prop, i);

        // NOTE: compute capability below 3.0 is not supported

        if (prop.major < 3) return;

        namesOfDevices.push_back(std::string(prop.name));

        auto globalMemInMB = prop.totalGlobalMem / (1024u * 1024u);

        globalMemoryOfDevices.push_back(static_cast<int32_t>(globalMemInMB));

        computeMajorCapabilityOfDevices.push_back(static_cast<int32_t>(prop.major));

        computeMinorCapabilityOfDevices.push_back(static_cast<int32_t>(prop.minor));
    }
#endif
}

}  // namespace gpu
