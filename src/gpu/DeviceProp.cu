#include "hip/hip_runtime.h"
//
//                     V.E.L.O.X. C.H.E.M. MP
//      ---------------------------------------------------
//           An Electronic Structure Code for Nanoscale
//
//  Created by Zilvinas Rinkevicius (rinkevic@kth.se), KTH, Sweden.
//  Copyright © 2018 by Velox Chem MP developers. All rights reserved.

#include <cmath>
#include <cstdio>

#include "DeviceProp.hpp"

namespace gpu { // gpu namespace

    void get_device_prop()
    {
        #ifdef ENABLE_GPU

        int nDevices = 0;

        hipGetDeviceCount(&nDevices);

        printf("Total Number of Devices: %d\n", nDevices);

        for (int i = 0; i < nDevices; i++) {

            hipDeviceProp_t prop;

            hipGetDeviceProperties(&prop, i);

            printf("Device ID: %d\n", i);

            printf("  Device name:                 %s\n", prop.name);

            printf("  Compute Capability:          %d.%d\n", prop.major, prop.minor);

            printf("  Number of Multiprocessors:   %d\n", prop.multiProcessorCount);

            printf("  GPU Max Clock Rate:          %.2f GHz\n", prop.clockRate * 1.0e-6);

            printf("  Global Memory on GPU:        %.0f GB\n", (float)prop.totalGlobalMem/pow(1024,3));

            printf("  Peak Memory Bandwidth:       %.0f GB/s\n",
                    2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e+6);

        }

        #endif
    }

}
